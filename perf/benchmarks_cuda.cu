#include <benchmark/benchmark.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <p2rng/bind.hpp>
#include <p2rng/pcg/pcg_random.hpp>
#include <p2rng/trng/uniform_dist.hpp>
#include <p2rng/algorithm/generate.hpp>

const unsigned long seed_pi{3141592654};

//----------------------------------------------------------------------------//
// generate() algortithm

template <class T>
void p2rng_generate_cuda(benchmark::State& st)
{   size_t n = size_t(st.range());
    hipEvent_t start, stop;
    hipEventCreate(&start); hipEventCreate(&stop);
    thrust::device_vector<T> v(n);

    for (auto _ : st)
    {   hipEventRecord(start);
        p2rng::generate
        (   v.begin()
        ,   v.end()
        ,   p2rng::bind(trng::uniform_dist<T>(10, 100), pcg32(seed_pi))
        );
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        st.SetIterationTime(milliseconds * 0.001f);
    }
    hipEventDestroy(start); hipEventDestroy(stop);

    st.counters["BW (GB/s)"] = benchmark::Counter
    (   (n * sizeof(T)) / 1e9
    ,   benchmark::Counter::kIsIterationInvariantRate
    );
}

BENCHMARK_TEMPLATE(p2rng_generate_cuda, float)
->  RangeMultiplier(2)
->  Range(1<<20, 1<<24)
->  UseManualTime()
->  Unit(benchmark::kMillisecond);

BENCHMARK_TEMPLATE(p2rng_generate_cuda, double)
->  RangeMultiplier(2)
->  Range(1<<20, 1<<24)
->  UseManualTime()
->  Unit(benchmark::kMillisecond);

//----------------------------------------------------------------------------//
// main()

int main(int argc, char** argv)
{   benchmark::Initialize(&argc, argv);
    if (benchmark::ReportUnrecognizedArguments(argc, argv))
        return 1;

    // adding GPU context
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::stringstream os;
    os << "\n  " << prop.name
       << "\n  L2 Cache: " << prop.l2CacheSize / 1024 << " KiB"
       << "\n  Number of SMs: x" << prop.multiProcessorCount
       << "\n  Peak Memory Bandwidth: "
       << std::fixed << std::setprecision(0)
       // based on https://developer.nvidia.com/blog/how-implement-performance-metrics-cuda-c
       << 2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6
       << " (GB/s)";
    benchmark::AddCustomContext("GPU", os.str());

    benchmark::RunSpecifiedBenchmarks();
    benchmark::Shutdown();
    return 0;
}